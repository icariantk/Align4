#include "hip/hip_runtime.h"
//Includes for IntelliSense
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <ctime>
#include  <stdio.h>
extern "C"
{
	__device__ int gap(int val){
		if (val == 0){ return 1; }
		if (val == 1){ return -4; }
		if (val == 2){ return -4; }
		return 0;
	}
	__global__ void align2(int *a, int *b, int *matrix, int *matrixDir, int *scoreMatrix, int m, int n)
	{
		extern __shared__ int order[];
		bool flag = 0;
		int index = blockIdx.x*blockDim.x + threadIdx.x;
		int x = threadIdx.x + 1;
		int topLeft, left, top;

		if (index == 0){
			for (int c = 0; c != m; c++){
				order[c] = 0;
			}
		}
		if (index < m){
			for (int c = 0; c != n; c++){
				matrix[c*m + index] = 0;
			}
		}

		__syncthreads();
		if (index < m - 1){
			for (int c = 0; c != n; c++)
			{
				if (index == 0)
				{
					matrixDir[0] = 0;
					matrixDir[c*m + index] = 3;
				}
				else
				{
					if (c == 0)
					{
						matrixDir[c*m + index] = 2;
					}
					else
					{
						matrixDir[c*m + index] = -1;
					}
				}
			}

			__syncthreads();

			for (int y = 1; y != n; y++){
				if (index == 0){
					order[0] = y;
					flag = 1;
				}
				else{
					if (order[index - 1] >= y){
						order[index] = y;
						flag = 1;
					}
					else{
						flag = 0;
					}
				}
				__syncthreads();

				if (flag){
					if ((a[index] != '-') && (b[y - 1] != '-')){
						topLeft = matrix[((y - 1)*m) + (x - 1)] + scoreMatrix[((a[x - 1] - 65) * 27) + (b[y - 1] - 65)];
					}
					else{
						topLeft = gap(1);
					}
					top = matrix[((y - 1)*m) + x] + gap(2);
					left = matrix[(y*m) + (x - 1)] + gap(2);

					if (topLeft >= left&&topLeft >= top){ matrixDir[y*m + x] = 1; matrix[y*m + x] = topLeft; }
					if (top > topLeft&&top >= left){ matrixDir[y*m + x] = 3; matrix[y*m + x] = top; }
					if (left > topLeft&&left > top){ matrixDir[y*m + x] = 2; matrix[y*m + x] = left; }
				}
				else{
					y--;
				}
				if (y == n - 1){
					order[index]++;
				}

				__syncthreads();
			}
		}
	}

	__global__ void traceback(int a[], int ai[], int b[], int bi[], int matrixDir[], int m, int n, int *k)
	{
		int x = m - 1;
		int y = n - 1;
		int c = 0;

		while (!(x == 0 && y == 0)){
			if (matrixDir[y*m + x] == 3){ ai[c] = '-'; if (y > 0){ bi[c] = b[y - 1]; y--; } }
			else
				if (matrixDir[y*m + x] == 2){ bi[c] = '-'; if (x > 0){ ai[c] = a[x - 1]; x--; } }
				else
					if (matrixDir[y*m + x] == 1){ if (x > 0 && y > 0){ ai[c] = a[x - 1]; bi[c] = b[y - 1]; x--; y--; } }
					else{
						x = 0; y = 0;
					}
					c++;
		}

		ai[c] = '\0';
		bi[c] = '\0';

		k[0] = c;
		k[0]--;
	}

	__global__ void invert(int a[], int ai[], int b[], int bi[], int k[])
	{
		int index = blockIdx.x*blockDim.x + threadIdx.x;
		if (index < k[0]){
			int vala = ai[k[0] - index - 1];
			int valb = bi[k[0] - index - 1];
			__syncthreads();
			a[index] = vala;
			b[index] = valb;
			__syncthreads();
		}
	}

	__global__ void alignPSP(int *a, int *b, int *matrix, int *matrixDir, int *scoreMatrix, int am, int an, int bm, int bn, int gap0, int gap1, int gap2, int offset, int size, int *order)
	{
		//gap0=gap gap, gap1=gap opening, gap2=gap mistmatch
		
		bool flag = 0;
		int index = (blockIdx.x*blockDim.x + threadIdx.x) + (offset * size);
		int x = index + 1;
		int topLeft, left, top;
		//printf("Soy el index: %d\n", index);
		//if (index == 0){
		//	printf("gaps inside cuda: %d %d %d\n", gap0, gap1, gap2);
		//}
		if (index < an+1){
			order[index] = 0;
			
				//matrix[c*(an + 1) + x] = 0;

				if (index == 0){
					for (int c = 0; c != bn + 1; c++){
						matrix[c*(an + 1)] = gap2*c;
					}
				} else{
					matrix[x] = gap2*x;
					matrixDir[x] = 2;
				}
			}

		

		__syncthreads();
		__threadfence_block();
		if (index < an+1){
			if (index == 0){
				for (int c = 0; c != bn + 1; c++)
				{
					matrixDir[c*(an + 1)] = 3;
				}
			}
			else{
				matrixDir[index] = 2;
			}
		}
		if (index == 0){
		matrixDir[0] = 0;
		}

		__syncthreads();
		__threadfence_block();
		if (index < an){
			//printf("\n%d 1\n", index);
			for (int y = 1; y <= bn; y++){
				if (index == 0){
					order[0] = y;
					flag = 1;
				}
				else{
					if (order[index - 1] > y){
						flag = 1;
					}
					else{
						flag = 0;
						y--;
					}
				}
				__syncthreads();
				__threadfence_block();

				if (flag){
					//	printf("\n%d 2\n", index);
					int sum = 0;
					for (int xx = 0; xx != am; xx++){
						for (int yy = 0; yy != bm; yy++){
							sum += (a[xx*an + (x - 1)] != '-'&&b[yy*bn + (y - 1)] != '-') ? scoreMatrix[((a[xx*an + (x - 1)] - 65) * 27) + (b[yy*bn + (y - 1)] - 65)] : a[xx*an + (x - 1)] == b[yy*bn + (y - 1)] ? gap0 : gap2;
							//	printf("Se compar�: %c y %c y salio: %d\n", a[xx*an + (x - 1)], b[yy*bn + (y - 1)], scoreMatrix[((a[xx*an + (x - 1)] - 65) * 27) + (b[yy*bn + (y - 1)] - 65)]);
						}
					}
					//Agregar el Gap OPENING

					topLeft = matrix[(y - 1)*(an + 1) + (x - 1)] + sum;

					sum = 0;
					for (int xx = 0; xx != am; xx++){
						for (int yy = 0; yy != bm; yy++){
							sum += a[xx*an + (x - 1)] == '-' ? gap0 : gap2;
						}
					}
					/*if (y > 1)
						sum += matrixDir[((y - 1)*(an + 1)) + x] == 1 ? gap1*am : 0;*/

					top = matrix[((y - 1)*(an + 1)) + x] + sum;

					sum = 0;
					for (int xx = 0; xx != am; xx++){
						for (int yy = 0; yy != bm; yy++){
							sum += b[yy*bn + (y - 1)] == '-' ? gap0 : gap2;
						}
					}
					/*if (x > 1)
						sum += matrixDir[y*(an + 1) + x - 1] == 1 ? gap1*bm : 0;*/

					left = matrix[(y*(an + 1)) + (x - 1)] + sum;

					//printf("matrix[%d,%d] top:%d topleft:%d left:%d\n", x, y, top, topLeft, left);
					if (topLeft >= left&&topLeft >= top){
						matrixDir[y*(an + 1) + x] = 1;
						matrix[y*(an + 1) + x] = topLeft;
						//	printf("matrix[%d,%d]=%d,%d\n", x, y, matrix[y*(an + 1) + x], matrixDir[y*(an + 1) + x]);
					}

					if (top > topLeft&&top >= left){
						matrixDir[y*(an + 1) + x] = 3;
						matrix[y*(an + 1) + x] = top;
						//printf("matrix[%d,%d]=%d,%d\n", x, y, matrix[y*(an + 1) + x], matrixDir[y*(an + 1) + x]);
					}

					if (left > topLeft&&left > top)
					{
						matrixDir[y*(an + 1) + x] = 2;
						matrix[y*(an + 1) + x] = left;
						//	printf("matrix[%d,%d]=%d,%d\n", x, y, matrix[y*(an + 1) + x], matrixDir[y*(an + 1) + x]);
					}
					//	printf("\n%d 3\n", index);
					/*if (index == 84&&y==90){
					printf("\nIndex 84 matrixDir[%d,%d]=%d\n", x, y, matrixDir[y*(an + 1) + x]);
					printf("matrix[%d,%d] top:%d topleft:%d left:%d\n", x, y, top, topLeft, left);
					}*/
					if (y == bn){
						order[index]++;
					}
					order[index]++;
				}
				__syncthreads();
			}
			//printf("\n%d 4\n", index);
			order[index]++;
			order[index]++;
			__syncthreads();
			__threadfence_block();
		}
	}

	__global__ void align2SIMO_Initialize(int *sizes, int x, int y, int sqrZone){
	}
	__global__ void kMerDistance(int *matrix, int *matrixDir, int *indexes, int *sequences, int *sizes, int nseq, int *scoreMatrix, int x, int y, int gap0, int gap1, int gap2, int *score, int sqrZone, int K)
	{
		//gap0=gap gap, gap1=gap opening, gap2=gap mistmatch

		int index = blockIdx.x*blockDim.x + threadIdx.x;
		int seqA = x*sqrZone + (index % sqrZone);
		bool flag;
		int seqB = y*sqrZone + (index / sqrZone);
		int sum = 0;
		int c, d, k;
		int *kmer;
		int sumA, sumB;
		if (x <= y){
			if (seqB < nseq&&seqA < nseq){
				int *A, *B;
				int offsetA = 0;
				int offsetB = 0;
				if (seqB > seqA){
					for (c = 0; c <= seqB; c++){
						if (c == seqA){ offsetA = sum; }
						if (c == seqB){ offsetB = sum; }
						sum += sizes[c];
					}
					A = &sequences[offsetA];
					B = &sequences[offsetB];
					for (d = 0; d != sizes[seqA] - K; d++){
						kmer = &A[d];

						for (c = 0; c != sizes[seqA] - K; c++){
							flag = true;
							for (k = 0; k != K; k++){
								if (kmer[k] != A[c + k]){
									flag = false;
									break;
								}
							}
							sumA += flag;
						}
						for (c = 0; c != sizes[seqB] - K; c++){
							flag = true;
							for (k = 0; k != K; k++){
								if (kmer[k] != B[c + k]){
									flag = false;
									break;
								}
							}
							sumB += flag;
						}
						if (sumA < sumB){
							sum += sumA;
						}
						else{
							sum += sumB;
						}
					}
					if (sizes[seqA] < sizes[seqB]){
						sum /= sizes[seqA] - K + 1;
					}
					else{
						sum /= sizes[seqB] - K + 1;
					}

					score[seqA*nseq + seqB] = sum;
					score[seqB*nseq + seqA] = sum;
				}
			}
		}
	}

	__global__ void align2SIMO(int *matrix, int *matrixDir, int *indexes, int *sequences, int *sizes, int nseq, int *scoreMatrix, int x, int y, int gap0, int gap1, int gap2, int *score, int sqrZone, int mode)
	{
		//gap0=gap gap, gap1=gap opening, gap2=gap mistmatch
		//mode  0=Column score 1=propossal
		int index = blockIdx.x*blockDim.x + threadIdx.x;
		int seqA = x*sqrZone + (index % sqrZone);
		int seqB = y*sqrZone + (index / sqrZone);
		//printf("\nholly\n");
		if (x <= y){
			/*		if (index == 0){
			matrix = new int**[(sqrZone*sqrZone)];
			matrixDir = new int**[(sqrZone*sqrZone)];
			for (int ccc = 0; ccc != sqrZone*sqrZone; ccc++){
			matrix[ccc] = new int*[sizes[x*sqrZone + (ccc % sqrZone)] + 1];
			matrixDir[ccc] = new int*[sizes[x*sqrZone + (ccc % sqrZone)] + 1];
			}
			for (int ccc = 0; ccc != sqrZone*sqrZone; ccc++){
			for (int d = 0; d != sizes[x*sqrZone + (ccc % sqrZone)] + 1; d++){
			matrix[ccc][d] = new int[sizes[y*sqrZone + (ccc / sqrZone)] + 1];
			matrixDir[ccc][d] = new int[sizes[y*sqrZone + (ccc / sqrZone)] + 1];
			}
			}
			}
			__syncthreads();
			*/
			//printf("\nholly\n");
			/**/
			if (seqB < nseq&&seqA < nseq){
				int offsetA = 0;
				int offsetB = 0;
				int sum = 0;
				int m = sizes[seqA];
				int n = sizes[seqB];
				/*int* A = new int[m];
				int* B = new int[n];*/
				int *A, *B;
				int *MatrixDir, *Matrix;
				int X, Y;
				if (seqB > seqA){
					for (int c = 0; c <= seqB; c++){
						if (c == seqA){ offsetA = sum; }
						if (c == seqB){ offsetB = sum; }
						sum += sizes[c];
					}
					//printf("\n2");

					int sSeqA = sizes[seqA] + 1;

					int offmatrix = indexes[index];
					Matrix = &matrix[offmatrix];
					MatrixDir = &matrixDir[offmatrix];

					for (X = 0; X != m + 1; X++){
						MatrixDir[X] = 2;
						Matrix[X] = 0;
					}
					for (Y = 0; Y != n + 1; Y++){
						MatrixDir[Y*sSeqA] = 3;
						Matrix[Y*sSeqA] = 0;
					}
					MatrixDir[0] = 0;
					Matrix[0] = 0;
					/*
					for (int x = 0; x != m + 1; x++){
					for (int y = 0; y != n + 1; y++){
					//printf("[%d](%d,%d) [ ]\n", blockIdx.x*blockDim.x + threadIdx.x, x, y, m, n);

					Matrix[ y*(sizes[seqA] + 1) + x] = 0;
					if (y == 0){
					MatrixDir[y*(sizes[seqA] + 1) + x] = 2;
					}
					else{
					MatrixDir[ y*(sizes[seqA] + 1) + x] = -1;
					}
					if (x == 0){
					MatrixDir[y*(sizes[seqA] + 1) + x] = 3;
					}
					if (x == 0 && y == 0){
					MatrixDir[y*(sizes[seqA] + 1) + x] = 0;
					}
					//printf("[%d](%d,%d) [X]\n", blockIdx.x*blockDim.x + threadIdx.x, x, y, m, n);
					}
					}
					*/

					//printf("\n3");
					A = &sequences[offsetA];
					B = &sequences[offsetB];

					/*
					for (int c = offsetA; c != offsetA + m; c++){
					A[c - offsetA] = sequences[c];
					}
					for (int c = offsetB; c != offsetB + n; c++){
					B[c - offsetB] = sequences[c];
					}*/

					//printf("\nFin de la inicializacion\n");
					for (int x = 1; x != m + 1; x++){
						for (int y = 1; y != n + 1; y++){
							int topLeft = 0, left = 0, top = 0;
							topLeft = Matrix[(y - 1)*(sSeqA)+(x - 1)] + scoreMatrix[((A[x - 1] - 65) * 27) + (B[y - 1] - 65)];
							top = Matrix[(y - 1)*(sSeqA)+(x)] - gap2;
							top += MatrixDir[(y - 1)*(sSeqA)+(x)] == 1 ? gap1 : 0;
							left = Matrix[(y)*(sSeqA)+(x - 1)] - gap2;
							left += MatrixDir[(y)*(sSeqA)+(x - 1)] == 1 ? gap1 : 0;
							if (topLeft >= left&&topLeft >= top){
								MatrixDir[(y)*(sSeqA)+(x)] = 1;
								Matrix[(y)*(sSeqA)+(x)] = topLeft;
							}
							if (top > topLeft&&top >= left){
								MatrixDir[(y)*(sSeqA)+(x)] = 3;
								Matrix[(y)*(sSeqA)+(x)] = top;
							}
							if (left > top&&left > topLeft)
							{
								MatrixDir[(y)*(sSeqA)+(x)] = 2;
								Matrix[(y)*(sSeqA)+(x)] = left;
							}
						}
					}

					//	printf("\nFin de valores a la matriz\n");
					X = m;
					Y = n;
					int scor = 0;

					while (!(X == 0 && Y == 0)){
						switch (MatrixDir[(Y)*(sSeqA)+(X)]){
						case 1: X--; Y--; break;
						case 2: X--; scor++; break;
						case 3: Y--; scor++; break;
						default: break;
						}
					}

					score[seqA*nseq + seqB] = scor;
					score[seqB*nseq + seqA] = scor;
					//	printf("\n[%d,%d]=%d\n",seqA,seqB,scor);
				}
				else{
					if (seqA == seqB)score[seqA*nseq + seqB] = 0;
				}
			}
		}
	}
	__global__ void align2SIMO_g(int *matrix, int *matrixDir, int *indexes, int *sequences, int *sizes, int nseq, int *scoreMatrix, int x, int y, int gap0, int gap1, int gap2, double *score, int sqrZone, int r)
	{
		//gap0=gap gap, gap1=gap opening, gap2=gap mistmatch
		//mode  0=Column score 1=propossal
		int index = blockIdx.x*blockDim.x + threadIdx.x;
		int seqA = x*sqrZone + (index % sqrZone);
		int seqB = y*sqrZone + (index / sqrZone);
		//printf("\nholly\n");
		if (x <= y){
			/*		if (index == 0){
			matrix = new int**[(sqrZone*sqrZone)];
			matrixDir = new int**[(sqrZone*sqrZone)];
			for (int ccc = 0; ccc != sqrZone*sqrZone; ccc++){
			matrix[ccc] = new int*[sizes[x*sqrZone + (ccc % sqrZone)] + 1];
			matrixDir[ccc] = new int*[sizes[x*sqrZone + (ccc % sqrZone)] + 1];
			}
			for (int ccc = 0; ccc != sqrZone*sqrZone; ccc++){
			for (int d = 0; d != sizes[x*sqrZone + (ccc % sqrZone)] + 1; d++){
			matrix[ccc][d] = new int[sizes[y*sqrZone + (ccc / sqrZone)] + 1];
			matrixDir[ccc][d] = new int[sizes[y*sqrZone + (ccc / sqrZone)] + 1];
			}
			}
			}
			__syncthreads();
			*/
			//printf("\nholly\n");
			/**/
			if (seqB < nseq&&seqA < nseq){
				int offsetA = 0;
				int offsetB = 0;
				int sum = 0;
				int m = sizes[seqA];
				int n = sizes[seqB];
				/*int* A = new int[m];
				int* B = new int[n];*/
				int *A, *B;
				int *MatrixDir, *Matrix;
				int X, Y;
				if (seqB > seqA){
					for (int c = 0; c <= seqB; c++){
						if (c == seqA){ offsetA = sum; }
						if (c == seqB){ offsetB = sum; }
						sum += sizes[c];
					}
					//printf("\n2");

					int sSeqA = sizes[seqA] + 1;
					int sSeqB = sizes[seqB];
					int offmatrix = indexes[index];
					Matrix = &matrix[offmatrix];
					MatrixDir = &matrixDir[offmatrix];

					for (X = 0; X != m + 1; X++){
						MatrixDir[X] = 2;
						Matrix[X] = 0;
					}
					for (Y = 0; Y != n + 1; Y++){
						MatrixDir[Y*sSeqA] = 3;
						Matrix[Y*sSeqA] = 0;
					}
					MatrixDir[0] = 0;
					Matrix[0] = 0;
					/*
					for (int x = 0; x != m + 1; x++){
					for (int y = 0; y != n + 1; y++){
					//printf("[%d](%d,%d) [ ]\n", blockIdx.x*blockDim.x + threadIdx.x, x, y, m, n);

					Matrix[ y*(sizes[seqA] + 1) + x] = 0;
					if (y == 0){
					MatrixDir[y*(sizes[seqA] + 1) + x] = 2;
					}
					else{
					MatrixDir[ y*(sizes[seqA] + 1) + x] = -1;
					}
					if (x == 0){
					MatrixDir[y*(sizes[seqA] + 1) + x] = 3;
					}
					if (x == 0 && y == 0){
					MatrixDir[y*(sizes[seqA] + 1) + x] = 0;
					}
					//printf("[%d](%d,%d) [X]\n", blockIdx.x*blockDim.x + threadIdx.x, x, y, m, n);
					}
					}
					*/

					//printf("\n3");
					A = &sequences[offsetA];
					B = &sequences[offsetB];
					for (int x = 1; x != m + 1; x++){
						for (int y = 1; y != n + 1; y++){
							int topLeft = 0, left = 0, top = 0;
							topLeft = Matrix[(y - 1)*(sSeqA)+(x - 1)] + scoreMatrix[((A[x - 1] - 65) * 27) + (B[y - 1] - 65)];
							top = Matrix[(y - 1)*(sSeqA)+(x)] + gap2;
							//top += MatrixDir[(y - 1)*(sSeqA)+(x)] == 1 ? gap1 : 0;
							left = Matrix[(y)*(sSeqA)+(x - 1)] + gap2;
							//left += MatrixDir[(y)*(sSeqA)+(x - 1)] == 1 ? gap1 : 0;
							if (topLeft >= left&&topLeft >= top){
								MatrixDir[(y)*(sSeqA)+(x)] = 1;
								Matrix[(y)*(sSeqA)+(x)] = topLeft;
							}
							if (top > topLeft&&top >= left){
								MatrixDir[(y)*(sSeqA)+(x)] = 3;
								Matrix[(y)*(sSeqA)+(x)] = top;
							}
							if (left > top&&left > topLeft)
							{
								MatrixDir[(y)*(sSeqA)+(x)] = 2;
								Matrix[(y)*(sSeqA)+(x)] = left;
							}
						}
					}

					//	printf("\nFin de valores a la matriz\n");
					X = m;
					Y = n;
					score[0] = 0;
					double  scor = 0;
					int *a_inv = new int[sSeqA + sSeqB];
					int *b_inv = new int[sSeqA + sSeqB];
					int size = 0;

					while (!(X == 0 && Y == 0)){
						switch (MatrixDir[(Y*sSeqA) + X]){
						case 1:	a_inv[size] = A[X - 1]; b_inv[size] = B[Y - 1]; X--; Y--; break;
						case 2: a_inv[size] = A[X - 1]; b_inv[size] = 45; X--; break;
						case 3: a_inv[size] = 45; b_inv[size] = B[Y - 1]; Y--; break;
						default: break;
						}
						size++;
					}

					for (int c = 0; c != size; c++){
						if (a_inv[c] == b_inv[c]){ scor++; }
					}
					score[seqA*nseq + seqB] = scor / (double)size;
					score[seqB*nseq + seqA] = scor / (double)size;

				}
				else{
					if (seqA == seqB)score[seqA*nseq + seqB] = 0;
				}
			}
		}
	}



	__global__ void align2SIMO_r(int *matrix, int *matrixDir, int *indexes, int *sequences, int *sizes, int nseq, int *scoreMatrix, int x, int y, int gap0, int gap1, int gap2, double *score, int sqrZone, int r)
	{
		//gap0=gap gap, gap1=gap opening, gap2=gap mistmatch
		//mode  0=Column score 1=propossal
		int index = blockIdx.x*blockDim.x + threadIdx.x;

		int seqA = x*sqrZone + (index % sqrZone);
		int seqB = y*sqrZone + (index / sqrZone);
		//printf("\nholly\n");
		if (x <= y){
			/*		if (index == 0){
			matrix = new int**[(sqrZone*sqrZone)];
			matrixDir = new int**[(sqrZone*sqrZone)];
			for (int ccc = 0; ccc != sqrZone*sqrZone; ccc++){
			matrix[ccc] = new int*[sizes[x*sqrZone + (ccc % sqrZone)] + 1];
			matrixDir[ccc] = new int*[sizes[x*sqrZone + (ccc % sqrZone)] + 1];
			}
			for (int ccc = 0; ccc != sqrZone*sqrZone; ccc++){
			for (int d = 0; d != sizes[x*sqrZone + (ccc % sqrZone)] + 1; d++){
			matrix[ccc][d] = new int[sizes[y*sqrZone + (ccc / sqrZone)] + 1];
			matrixDir[ccc][d] = new int[sizes[y*sqrZone + (ccc / sqrZone)] + 1];
			}
			}
			}
			__syncthreads();
			*/
			
			/**/
			
			if (seqB < nseq&&seqA < nseq){
				
				int offsetA = 0;
				int offsetB = 0;
				int sum = 0;
				int m = sizes[seqA];
				int n = sizes[seqB];
				/*int* A = new int[m];
				int* B = new int[n];*/
				int *A, *B;
				int *MatrixDir, *Matrix;
				int X, Y;
				if (seqB > seqA){
					for (int c = 0; c <= seqB; c++){
						if (c == seqA){ offsetA = sum; }
						if (c == seqB){ offsetB = sum; }
						sum += sizes[c];
					}
					

					int sSeqA = sizes[seqA] + 1;
					int sSeqB = sizes[seqB];
					int offmatrix = indexes[index];
					Matrix = &matrix[offmatrix];
					MatrixDir = &matrixDir[offmatrix];

					for (X = 0; X != m + 1; X++){
						MatrixDir[X] = 2;
						Matrix[X] = 0;
					}
					for (Y = 0; Y != n + 1; Y++){
						MatrixDir[Y*sSeqA] = 3;
						Matrix[Y*sSeqA] = 0;
					}
					MatrixDir[0] = 0;
					Matrix[0] = 0;
					/*
					for (int x = 0; x != m + 1; x++){
					for (int y = 0; y != n + 1; y++){
					//printf("[%d](%d,%d) [ ]\n", blockIdx.x*blockDim.x + threadIdx.x, x, y, m, n);

					Matrix[ y*(sizes[seqA] + 1) + x] = 0;
					if (y == 0){
					MatrixDir[y*(sizes[seqA] + 1) + x] = 2;
					}
					else{
					MatrixDir[ y*(sizes[seqA] + 1) + x] = -1;
					}
					if (x == 0){
					MatrixDir[y*(sizes[seqA] + 1) + x] = 3;
					}
					if (x == 0 && y == 0){
					MatrixDir[y*(sizes[seqA] + 1) + x] = 0;
					}
					//printf("[%d](%d,%d) [X]\n", blockIdx.x*blockDim.x + threadIdx.x, x, y, m, n);
					}
					}
					*/
					/*
					if (seqA == 0 && seqB == 1){
					printf("Blosum:\n");
					for (int c = 0; c != 10; c++){
						printf("%d ", scoreMatrix[c]);
					}
					}*/
					
					A = &sequences[offsetA];
					B = &sequences[offsetB];

					for (int x = 1; x != m + 1; x++){
						for (int y = 1; y != n + 1; y++){
							int topLeft = 0, left = 0, top = 0;
							topLeft = Matrix[(y - 1)*(sSeqA)+(x - 1)] + scoreMatrix[((A[x - 1] - 65) * 27) + (B[y - 1] - 65)];
							top = Matrix[(y - 1)*(sSeqA)+(x)] + gap2;
							top += MatrixDir[(y - 1)*(sSeqA)+(x)] == 1 ? gap1 : 0;
							left = Matrix[(y)*(sSeqA)+(x - 1)] + gap2;
							left += MatrixDir[(y)*(sSeqA)+(x - 1)] == 1 ? gap1 : 0;
							if (topLeft >= left&&topLeft >= top){
								MatrixDir[(y)*(sSeqA)+(x)] = 1;
								Matrix[(y)*(sSeqA)+(x)] = topLeft;
							}
							if (top > topLeft&&top >= left){
								MatrixDir[(y)*(sSeqA)+(x)] = 3;
								Matrix[(y)*(sSeqA)+(x)] = top;
							}
							if (left > top&&left > topLeft)
							{
								MatrixDir[(y)*(sSeqA)+(x)] = 2;
								Matrix[(y)*(sSeqA)+(x)] = left;
							}
						}
					}

					
					X = m;
					Y = n;
					score[0] = 0;
					double  scor = 0;
					int *a_inv = new int[sSeqA + sSeqB];
					int *b_inv = new int[sSeqA + sSeqB];
					int size = 0;

					while (!(X == 0 && Y == 0)){
						switch (MatrixDir[(Y*sSeqA) + X]){
						case 1:	a_inv[size] = A[X - 1]; b_inv[size] = B[Y - 1]; X--; Y--; break;
						case 2: a_inv[size] = A[X - 1]; b_inv[size] = 45; X--; break;
						case 3: a_inv[size] = 45; b_inv[size] = B[Y - 1]; Y--; break;
						
						}
						size++;
					}
					double g=0, h=0;
					for (int c = 0; c != size; c++){
						for (int d = c - r; d != c + r; d++){
							if (d >= 0 && d < size){
								if (d < c){
									
										if (a_inv[c] != '-'&&b_inv[d] != '-'){
											g = (double)scoreMatrix[((a_inv[c] - 65) * 27) + (b_inv[d] - 65)];
										}
										else{
											g = (double)gap0;
										}
										g += 4 - gap1;
										h = r - (d - (c - r)) + 1.0;

										scor += (h / g);
									
								}
								if (d == c){
									if (a_inv[c] != '-'&&b_inv[d] != '-'){
										scor += (double)scoreMatrix[((a_inv[c] - 65) * 27) + (b_inv[d] - 65)];
									}
									else{
										scor += gap0;
									}
								}
								if (d > c){
									if (d < size){
										if (a_inv[c] != '-'&&b_inv[d] != '-'){
											g = (double)scoreMatrix[((a_inv[c] - 65) * 27) + (b_inv[d] - 65)];
										}
										else{
											g = gap0;
										}
										g += 4 - gap1;
										h = (d - (c - r)) + 1.0;
										scor += (h / g);
									}
								}
							}
						}
					}

					score[seqA*nseq + seqB] = scor;
					score[seqB*nseq + seqA] = scor;
					/*
					printf("\n[%d,%d] size %d score:%f\n", m, n, size, scor);
					if (seqA == 0 && seqB == 1){
					for (int c = 0; c != size; c++){
						printf("%c", a_inv[c]);
					}
					
						printf("\n");
					
					for (int c = 0; c != size; c++){
						printf("%c", b_inv[c]);
					}
					printf("\n");
					}*/
				}
				else{
					if (seqA == seqB)score[seqA*nseq + seqB] = 0;
				}
			}
		}
	}

	__global__ void tracebackPSP(int inv[], int a[], int b[], int matrixDir[], int am, int an, int bm, int bn, int *k)
	{
		int x = an;
		int y = bn;
		int c = 0;

		int maxInv = (an + bn);
		for (int k = 0; k != maxInv*(am + bm); k++){
			inv[k] = 64;
		}

		while (!(x == 0 && y == 0)){
			if (matrixDir[y*(an + 1) + x] == 3){
				for (int d = 0; d != am; d++){
					inv[(d*maxInv) + c] = '-';
				}
				for (int d = am; d != am + bm; d++){
					inv[((d*maxInv) + c)] = b[(d - am)*bn + y - 1];
				}
				y--;
			}
			else
			{
				if (matrixDir[y*(an + 1) + x] == 2)
				{
					for (int d = 0; d != am; d++){
						inv[(d*maxInv) + c] = a[d*an + x - 1];
					}
					for (int d = am; d != am + bm; d++){
						inv[((d*maxInv) + c)] = '-';
					}
					x--;
				}
				else
				{
					if (matrixDir[y*(an + 1) + x] == 1)
					{
						for (int d = 0; d != am; d++){
							inv[(d*maxInv) + c] = a[d*an + x - 1];
						}
						for (int d = am; d != am + bm; d++){
							inv[((d*maxInv) + c)] = b[(d - am)*bn + y - 1];
						}

						x--;
						y--;
					}

				}
			}

			c++;
		}

		k[0] = c;
	}
	__global__ void invertPSP(int original[], int inverse[], int n, int k[], int mn)
	{
		for (int d = 0; d != n; d++){
			for (int c = 0; c < k[0]; c++){
				if (inverse[(mn*d) + (k[0] - c - 1)] > 0 && inverse[(mn*d) + (k[0] - c)] < 256){
					original[(mn*d) + c] = inverse[(mn*d) + (k[0] - c - 1)];
				}
				else{
					original[(mn*d) + c] = 32;
				}
			}
		}
	}
	int main(){
		return 0;
	}
}

/*  6 febrero 2015
__global__ void alignPSP(int *a, int *b, int *matrix, int *matrixDir, int *scoreMatrix, int am, int an, int bm, int bn, int gap0, int gap1, int gap2, int offset, int size, int *order)
{
//gap0=gap gap, gap1=gap opening, gap2=gap mistmatch

bool flag = 0;
int index = (blockIdx.x*blockDim.x + threadIdx.x) + (offset * size);
int x = index + 1;
int topLeft, left, top;
printf("Soy el index: %d\n",index);
if (index <= an){
order[index] = 0;
for (int c = 0; c != bn + 1; c++){
matrix[c*(an + 1) + x] = 0;
matrixDir[c*(an + 1) + x] = 0;
}
matrixDir[x] = 2;
}

__syncthreads();
__threadfence_block();
if (index <= an){
if (index != 0 && offset == 0){
for (int c = 0; c != bn + 1; c++)
{
matrixDir[c*(an + 1) + index] = c == 0 ? 2 : 5;
}
}
else{
for (int c = 0; c != bn + 1; c++)
{
matrixDir[c*(an + 1)] = 3;
}
matrixDir[0] = 0;
}
}

__syncthreads();
__threadfence_block();
if (index < an){
for (int y = 1; y <= bn; y++){
if (index == 0){
order[0] = y;
flag = 1;
}
else{
//printf("Yo: %d order[%d]=%d\n", index,index-1,order[index-1]);
if (order[index - 1] > y){
flag = 1;
order[index]++;
}
else{
flag = 0;
y--;
}
}
__syncthreads();
__threadfence_block();

if (flag){
int sum = 0;
for (int xx = 0; xx != am; xx++){
for (int yy = 0; yy != bm; yy++){
if ((a[xx*an + (x - 1)] != '-'&&b[yy*bn + (y - 1)] != '-')){
sum += scoreMatrix[((a[xx*an + (x - 1)] - 65) * 27) + (b[yy*bn + (y - 1)] - 65)];
//	printf("Se compar�: %c y %c y salio: %d\n", a[xx*an + (x - 1)], b[yy*bn + (y - 1)], scoreMatrix[((a[xx*an + (x - 1)] - 65) * 27) + (b[yy*bn + (y - 1)] - 65)]);
}
else{
if (a[xx*an + (x - 1)] == b[yy*bn + (y - 1)]){
sum += gap0;
}
else{
sum += gap2;
}
}
}
}

topLeft = matrix[(y - 1)*(an+1) + (x - 1)] + sum;

sum = 0;
for (int xx = 0; xx != am; xx++){
for (int yy = 0; yy != bm; yy++){
if (a[xx*an + (x - 1)] == '-'){
sum += gap0;
}
else{
sum += gap2;
}
}
}
if (y>1)
sum += matrixDir[((y-1)*(an+1)) + x] == 1 ? gap1 : 0;

top = matrix[((y - 1)*(an+1)) + x] + sum;
sum = 0;
for (int xx = 0; xx != am; xx++){
for (int yy = 0; yy != bm; yy++){
if (b[yy*bn + (y - 1)] == '-'){
sum += gap0;
}
else{
sum += gap2;
}
}
}
if (x>1)
sum += matrixDir[y*(an + 1) + x-1] == 1 ? gap1 : 0;
left = matrix[(y*(an+1)) + (x - 1)] + sum;
//printf("matrix[%d,%d] top:%d topleft:%d left:%d\n", x, y, top, topLeft, left);
if (topLeft >= left&&topLeft >= top){
matrixDir[y*(an + 1) + x] = 1;
matrix[y*(an + 1) + x] = topLeft;
//	printf("matrix[%d,%d]=%d,%d\n", x, y, matrix[y*(an + 1) + x], matrixDir[y*(an + 1) + x]);
}
else
if (top > topLeft&&top >= left){
matrixDir[y*(an + 1) + x] = 3;
matrix[y*(an + 1) + x] = top;
//printf("matrix[%d,%d]=%d,%d\n", x, y, matrix[y*(an + 1) + x], matrixDir[y*(an + 1) + x]);
}
else
{
matrixDir[y*(an + 1) + x] = 2;
matrix[y*(an + 1) + x] = left;
//	printf("matrix[%d,%d]=%d,%d\n", x, y, matrix[y*(an + 1) + x], matrixDir[y*(an + 1) + x]);
}
if (y == bn){
order[index]++;
}
}
}

order[index]++;
order[index]++;
__syncthreads();
__threadfence_block();
}
}

*/